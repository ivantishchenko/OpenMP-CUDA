#include "hip/hip_runtime.h"
#include "cp.h"
#include <hip/hip_runtime.h>
#include <iostream>

template<typename T>
hipError_t allocDev(T*& d_p, size_t elements) {
    return hipMalloc((void**)&d_p, elements * sizeof(T));
}

template<typename T>
hipError_t allocHost(T*& d_p, size_t elements) {
    return hipHostAlloc((void**)&d_p, elements * sizeof(T), hipHostMallocMapped);
}

__global__ void medianKernel(float * output, const float * input, int ny, int nx) {
    int x = threadIdx.x + blockIdx.x * blockDim.x; 
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= ny || y >= ny) return;
    
    //int index = x + nx * y; 
    //int res = index + 1;   
    //output[index] = res; 
    /// CORELATE
 
 
    if (y > x) {
        return;
    }
    float sab, a, b;

            sab = 0.0;
            for (int j = 0; j < nx; j++) {
                a = input[x * nx + j];
                b = input[y * nx + j];
                sab += a * b;
                //printf("a = %d ; b = %d\n", x, y);
            }
            
            output[x + y * ny] = sab;
        
            //printf("%d  ", sab);
    //CORELaTE
    
    
}

void correlate(int ny, int nx, const float* data, float* result) {
    const size_t N = nx * ny;      
    float * input_CPU = nullptr;
   // double * output_CPU = nullptr;  
    float * input_GPU = nullptr;  
    float * output_GPU = nullptr;  
    
    // ALOC CPU
    hipError_t res_in_host = allocHost(input_CPU, N);      // modifies d, input is not bytes
    //hipError_t res_out_host = allocHost(output_CPU, N);      // modifies d, input is not bytes
    // ALOC GPU
    hipError_t res_in_dev = allocDev(input_GPU, N);     
    hipError_t res_out_dev = allocDev(output_GPU, ny * ny); 
    //Check for errors
  //  if (res_in_host == hipSuccess && res_out_host == hipSuccess ) std::cout << "Allocated host memory" << std::endl;
   // if (res_in_dev == hipSuccess && res_out_dev == hipSuccess ) std::cout << "Allocated host memory" << std::endl;

    //OPTIMAZATION
    
    float sum, sum_sqr;

    // first optimization
    for ( int i = 0; i < ny; i++ ) {
        sum = 0;
        for ( int j = 0; j < nx; j++ ) {
            sum += data[j + i * nx];
        }
        sum = sum / nx;
        sum_sqr = 0;
        for ( int j = 0; j < nx; j++ ) {
            input_CPU[j + i * nx] = data[j + i * nx] - sum;
            sum_sqr += input_CPU[j + i * nx] * input_CPU[j + i * nx];
        }
        sum_sqr = std::sqrt(sum_sqr);
        for ( int j = 0; j < nx; j++ ) {
            input_CPU[j + i * nx] /= sum_sqr;            
        }
    }
    
  
        // CPU -> GPU
    hipMemcpy(input_GPU, input_CPU, N * sizeof(float), hipMemcpyHostToDevice);
    
    // block size
    dim3 dimBlock(8, 8);
    dim3 dimGrid((ny + dimBlock.x - 1) / dimBlock.x, (ny + dimBlock.y - 1) / dimBlock.y);
    //launch KERNEL
    medianKernel<<<dimGrid, dimBlock>>>(output_GPU, input_GPU, ny, nx);
    // GPU -> CPU
    hipMemcpy(result, output_GPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost);
    
    
    
    ///std::cout << std::endl << "HELLO WORLD    ["<<output_CPU[3] << "] :from the GPU" << std::endl;
    
    hipFree(input_CPU);
    //hipFree(output_CPU);
    hipFree(input_GPU);
    hipFree(output_GPU);
}